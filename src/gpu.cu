#include "gpu.h"


#ifdef __cplusplus
extern "C" {
#endif

void gpuAssert(hipError_t code, const char *file, int line)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (1) exit(code);
   }
}

#ifdef __cplusplus
}
#endif
